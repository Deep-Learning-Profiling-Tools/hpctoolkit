#include <hip/hip_runtime.h>
#include <iostream>

__device__ static void func1(volatile double* x) {
  for(unsigned int i = 0; i < 1; i++)
    *x = *x * 2 + 3;
  *x = *x * 2 + 3;
}

__device__ static void func2(volatile double* x) {
  for(unsigned int i = 0; i < 1; i++) {
    for(unsigned int j = 0; j < 1; j++)
      *x = *x * 2 + 3;
    *x = *x * 2 + 3;
  }
  *x = *x * 2 + 3;
}

__global__ static void kernmain() {
  volatile double x = 2;
  for(unsigned int i = 0; i < 1<<12; i++) {
    for(unsigned int j = 0; j < 1; j++) {
      for(unsigned int k = 0; k < 1; k++)
        x = x * 2 + 3;
      x = x * 2 + 3;
      func1(&x);
    }
    for(unsigned int k = 0; k < 1; k++)
      x = x * 2 + 3;
    x = x * 2 + 3;
    func2(&x);
  }
}

int main() {
  hipError_t err;

  // Check that we have a device to work on
  {
    int nDevices = 0;
    err = hipGetDeviceCount(&nDevices);
    if(err != hipSuccess || nDevices == 0) {
      std::cerr << "No devices available!\n";
      return 77;  // SKIP
    }
  }

  for(int i = 0; i < 100; i++) {
    // Launch the kernel
    kernmain<<<100, 32>>>();
    err = hipGetLastError();
    if(err != hipSuccess) {
      std::cerr << "Error during kernel launch\n";
      return 1;
    }

    // Wait for the kernel to complete
    err = hipDeviceSynchronize();
    if(err != hipSuccess) {
      std::cerr << "Error returned by kernel\n";
      return 1;
    }
  }

  return 0;
}
